#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/detail/raw_pointer_cast.h>
#include <cudf/ast/ast.cuh>
#include <cudf/ast/linearizer.cuh>
#include <cudf/ast/operators.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <functional>
#include <iterator>
#include <type_traits>

namespace cudf {

namespace ast {

namespace detail {

template <typename Element>
__device__ Element row_evaluator::resolve_input(
  const detail::device_data_reference device_data_reference, cudf::size_type row_index) const
{
  auto const data_index = device_data_reference.data_index;
  switch (device_data_reference.reference_type) {
    case detail::device_data_reference_type::COLUMN: {
      auto column = this->table.column(data_index);
      return column.data<Element>()[row_index];
    }
    case detail::device_data_reference_type::LITERAL: {
      return this->literals[data_index].value<Element>();
    }
    case detail::device_data_reference_type::INTERMEDIATE: {
      return *reinterpret_cast<const Element*>(&this->thread_intermediate_storage[data_index]);
    }
    default: {
      release_assert(false && "Invalid input device data reference type.");
      return Element();
    }
  }
}

template <typename Element>
__device__ Element* row_evaluator::resolve_output(
  const detail::device_data_reference device_data_reference, cudf::size_type row_index) const
{
  switch (device_data_reference.reference_type) {
    case detail::device_data_reference_type::COLUMN: {
      // TODO: Could refactor to support output tables (multiple output columns)
      return &(this->output_column->element<Element>(row_index));
    }
    case detail::device_data_reference_type::INTERMEDIATE: {
      return reinterpret_cast<Element*>(
        &this->thread_intermediate_storage[device_data_reference.data_index]);
    }
    default: {
      release_assert(false && "Invalid output device data reference type.");
      return nullptr;
    }
  }
}

template <typename OperatorFunctor,
          typename Input,
          typename Out,
          std::enable_if_t<cudf::ast::is_valid_unary_op<OperatorFunctor, Input>>*>
__device__ void row_evaluator::operator()(cudf::size_type row_index,
                                          const detail::device_data_reference input,
                                          const detail::device_data_reference output) const
{
  auto const typed_input = this->resolve_input<Input>(input, row_index);
  auto typed_output      = this->resolve_output<Out>(output, row_index);
  *typed_output          = OperatorFunctor{}(typed_input);
}

template <typename OperatorFunctor,
          typename Input,
          typename Out,
          std::enable_if_t<!cudf::ast::is_valid_unary_op<OperatorFunctor, Input>>*>
__device__ void row_evaluator::operator()(cudf::size_type row_index,
                                          const detail::device_data_reference input,
                                          const detail::device_data_reference output) const
{
  release_assert(false && "Invalid unary dispatch operator for the provided input.");
}

template <typename OperatorFunctor,
          typename LHS,
          typename RHS,
          typename Out,
          std::enable_if_t<cudf::ast::is_valid_binary_op<OperatorFunctor, LHS, RHS>>*>
__device__ void row_evaluator::operator()(cudf::size_type row_index,
                                          const detail::device_data_reference lhs,
                                          const detail::device_data_reference rhs,
                                          const detail::device_data_reference output) const
{
  auto const typed_lhs = this->resolve_input<LHS>(lhs, row_index);
  auto const typed_rhs = this->resolve_input<RHS>(rhs, row_index);
  auto typed_output    = this->resolve_output<Out>(output, row_index);
  *typed_output        = OperatorFunctor{}(typed_lhs, typed_rhs);
}

template <typename OperatorFunctor,
          typename LHS,
          typename RHS,
          typename Out,
          std::enable_if_t<!cudf::ast::is_valid_binary_op<OperatorFunctor, LHS, RHS>>*>
__device__ void row_evaluator::operator()(cudf::size_type row_index,
                                          const detail::device_data_reference lhs,
                                          const detail::device_data_reference rhs,
                                          const detail::device_data_reference output) const
{
  release_assert(false && "Invalid binary dispatch operator for the provided input.");
}

__device__ void evaluate_row_expression(const detail::row_evaluator evaluator,
                                        const detail::device_data_reference* data_references,
                                        const ast_operator* operators,
                                        const cudf::size_type* operator_source_indices,
                                        cudf::size_type num_operators,
                                        cudf::size_type row_index)
{
  auto operator_source_index = cudf::size_type(0);
  for (cudf::size_type operator_index(0); operator_index < num_operators; operator_index++) {
    // Execute operator
    auto const op    = operators[operator_index];
    auto const arity = cudf::ast::ast_operator_arity(op);
    if (arity == 1) {
      // Unary operator
      auto const input  = data_references[operator_source_indices[operator_source_index]];
      auto const output = data_references[operator_source_indices[operator_source_index + 1]];

      unary_operator_dispatcher(op, input.data_type, evaluator, row_index, input, output);
    } else if (arity == 2) {
      // Binary operator
      auto const lhs    = data_references[operator_source_indices[operator_source_index]];
      auto const rhs    = data_references[operator_source_indices[operator_source_index + 1]];
      auto const output = data_references[operator_source_indices[operator_source_index + 2]];
      binary_operator_dispatcher(
        op, lhs.data_type, rhs.data_type, evaluator, row_index, lhs, rhs, output);
    } else {
      release_assert(false && "Invalid operator arity.");
      // Ternary operator
      /*
      auto const condition_data_ref =
        data_references[operator_source_indices[operator_source_index]];
      auto const lhs_data_ref = data_references[operator_source_indices[operator_source_index + 1]];
      auto const rhs_data_ref = data_references[operator_source_indices[operator_source_index + 2]];
      auto const output_data_ref =
        data_references[operator_source_indices[operator_source_index + 3]];
      operate(op,
              table,
              output_column,
              literals,
              thread_intermediate_storage,
              row_index,
              condition_data_ref,
              lhs_data_ref,
              rhs_data_ref,
              output_data_ref);
      */
    }
    operator_source_index += (arity + 1);
  }
}

template <size_type block_size>
__launch_bounds__(block_size) __global__
  void compute_column_kernel(const table_device_view table,
                             const cudf::detail::fixed_width_scalar_device_view_base* literals,
                             mutable_column_device_view output_column,
                             const detail::device_data_reference* data_references,
                             const ast_operator* operators,
                             const cudf::size_type* operator_source_indices,
                             cudf::size_type num_operators,
                             cudf::size_type num_intermediates)
{
  extern __shared__ std::int64_t intermediate_storage[];
  auto thread_intermediate_storage = &intermediate_storage[threadIdx.x * num_intermediates];
  const cudf::size_type start_idx  = threadIdx.x + blockIdx.x * blockDim.x;
  const cudf::size_type stride     = blockDim.x * gridDim.x;
  auto const num_rows              = table.num_rows();
  auto const evaluator =
    cudf::ast::detail::row_evaluator(table, literals, thread_intermediate_storage, &output_column);

  for (cudf::size_type row_index = start_idx; row_index < num_rows; row_index += stride) {
    evaluate_row_expression(
      evaluator, data_references, operators, operator_source_indices, num_operators, row_index);
  }
}

template <typename T>
rmm::device_uvector<T> async_create_device_data(std::vector<T> host_data, hipStream_t stream)
{
  auto device_data = rmm::device_uvector<T>(host_data.size(), stream);
  CUDA_TRY(hipMemcpyAsync(device_data.data(),
                           host_data.data(),
                           sizeof(T) * host_data.size(),
                           hipMemcpyHostToDevice,
                           stream));
  return device_data;
}

std::unique_ptr<column> compute_column(table_view const table,
                                       std::reference_wrapper<const expression> expr,
                                       hipStream_t stream,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  // Linearize the AST
  nvtxRangePush("Linearizing...");
  auto expr_linearizer = linearizer(table);
  expr.get().accept(expr_linearizer);
  auto const data_references         = expr_linearizer.get_data_references();
  auto const literals                = expr_linearizer.get_literals();
  auto const operators               = expr_linearizer.get_operators();
  auto const num_operators           = cudf::size_type(operators.size());
  auto const operator_source_indices = expr_linearizer.get_operator_source_indices();
  auto const expr_data_type          = expr_linearizer.get_root_data_type();
  nvtxRangePop();

  // Create device data
  nvtxRangePush("Creating device data...");
  auto const device_data_references = detail::async_create_device_data(data_references, stream);
  auto const device_literals        = detail::async_create_device_data(literals, stream);
  auto const device_operators       = detail::async_create_device_data(operators, stream);
  auto const device_operator_source_indices =
    detail::async_create_device_data(operator_source_indices, stream);
  // The stream is synced later when the table_device_view is created.
  // To reduce overhead, we don't call a stream sync here.
  nvtxRangePop();

  // Output linearizer info
  /*
  std::cout << "LINEARIZER INFO:" << std::endl;
  std::cout << "Number of data references: " << data_references.size() << std::endl;
  std::cout << "Data references: ";
  for (auto const& dr : data_references) {
    switch (dr.reference_type) {
      case detail::device_data_reference_type::COLUMN: std::cout << "C"; break;
      case detail::device_data_reference_type::LITERAL: std::cout << "L"; break;
      case detail::device_data_reference_type::INTERMEDIATE: std::cout << "I";
    }
    std::cout << dr.data_index << ", ";
  }
  std::cout << std::endl;
  std::cout << "Number of operators: " << num_operators << std::endl;
  std::cout << "Number of operator source indices: " << operator_source_indices.size() << std::endl;
  std::cout << "Number of literals: " << literals.size() << std::endl;
  std::cout << "Operator source indices: ";
  for (auto const& v : operator_source_indices) { std::cout << v << ", "; }
  std::cout << std::endl;
  */

  // Create table device view
  nvtxRangePush("Creating table device view...");
  auto table_device         = table_device_view::create(table, stream);
  auto const table_num_rows = table.num_rows();
  nvtxRangePop();

  // Prepare output column
  nvtxRangePush("Preparing output column...");
  auto output_column = cudf::make_fixed_width_column(
    expr_data_type, table_num_rows, mask_state::UNALLOCATED, stream, mr);
  auto mutable_output_device =
    cudf::mutable_column_device_view::create(output_column->mutable_view(), stream);
  nvtxRangePop();

  // Configure kernel parameters
  nvtxRangePush("Configuring kernel parameters...");
  auto constexpr block_size = 512;
  cudf::detail::grid_1d config(table_num_rows, block_size);
  auto const num_intermediates = expr_linearizer.get_intermediate_count();
  auto const shmem_size_per_block =
    sizeof(std::int64_t) * num_intermediates * config.num_threads_per_block;
  /*
  std::cout << "Requesting " << config.num_blocks << " blocks, ";
  std::cout << config.num_threads_per_block << " threads/block, ";
  std::cout << shmem_size_per_block << " bytes of shared memory." << std::endl;
  */
  nvtxRangePop();

  // Execute the kernel
  nvtxRangePush("Executing AST kernel...");
  cudf::ast::detail::compute_column_kernel<block_size>
    <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream>>>(
      *table_device,
      thrust::raw_pointer_cast(device_literals.data()),
      *mutable_output_device,
      thrust::raw_pointer_cast(device_data_references.data()),
      thrust::raw_pointer_cast(device_operators.data()),
      thrust::raw_pointer_cast(device_operator_source_indices.data()),
      num_operators,
      num_intermediates);
  CHECK_CUDA(stream);
  nvtxRangePop();
  return output_column;
}

}  // namespace detail

std::unique_ptr<column> compute_column(table_view const table,
                                       std::reference_wrapper<const expression> expr,
                                       rmm::mr::device_memory_resource* mr)
{
  return detail::compute_column(table, expr, 0, mr);
}

}  // namespace ast

}  // namespace cudf